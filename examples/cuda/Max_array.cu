
#include <hip/hip_runtime.h>
#define MAX(x, y)((x > y) ? x : y)

__global__ void find_max(const int * array, int * max, int * mutex, unsigned int n) {
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int stride = gridDim.x * blockDim.x;
    unsigned int input_idx = idx;

    __shared__ int partial_res[256];

    int local_max = array[0];
    while (input_idx < n) {
        local_max = MAX(local_max, array[input_idx]);
        input_idx += stride;
    }

    partial_res[threadIdx.x] = local_max;

    __syncthreads();

    // reduction
    unsigned int i = blockDim.x / 2;
    while (i != 0) {
        if (threadIdx.x < i) {
            partial_res[threadIdx.x] = MAX(partial_res[threadIdx.x], partial_res[threadIdx.x + i]);
        }

        __syncthreads();
        i /= 2;
    }

    if (threadIdx.x == 0) {
        while (atomicCAS(mutex, 0, 1) != 0);
        * max = MAX( * max, partial_res[0]);
        atomicExch(mutex, 0);
    }
}
