
#include <hip/hip_runtime.h>
__global__ void sum_array(const int * array, int * total, unsigned int n) {
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int stride = gridDim.x * blockDim.x;
    unsigned int input_idx = idx;

    __shared__ int partial_res[256];

    int partial_sum = 0;
    while (input_idx < n) {
        partial_sum += array[input_idx];
        input_idx += stride;
    }

    partial_res[threadIdx.x] = partial_sum;

    __syncthreads();

    // reduction
    unsigned int i = blockDim.x / 2;
    while (i != 0) {
        if (threadIdx.x < i) {
            partial_res[threadIdx.x] += partial_res[threadIdx.x + i];
        }
        __syncthreads();
        i /= 2;
    }

    if (threadIdx.x == 0) {
        atomicAdd(total, partial_res[0]);
    }
}
