
#include <hip/hip_runtime.h>
// Type your code here, or load an example.
__global__ void square(int* array, int n) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < n)
        array[tid] = array[tid] * array[tid];
}
